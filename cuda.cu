// cuda.cu – Basic CUDA example


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloKernel() {
    printf("Hello from GPU thread %d\n", threadIdx.x);
}

int main() {
    // Launch 5 threads in one block
    helloKernel<<<1,5>>>();
    hipDeviceSynchronize();       // Wait for GPU to finish
    return 0;
}
